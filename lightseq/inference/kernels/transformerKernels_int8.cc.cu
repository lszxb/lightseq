#include "hip/hip_runtime.h"
#include "common.h"
#include "transformerKernels.h"
#include "transformerKernels_int8.h"

/**
@file
Implemented the cuda kernel function and its launcher
that required by transformer model.
Currently, fp16 and fp32 versions are provided
*/
namespace lightseq {
namespace cuda {
__forceinline__ __host__ __device__ int8_t float2int8(float x,
                                                      float scale_div_clip_max,
                                                      float clip_max) {
  x = x > clip_max ? clip_max : (x < -clip_max ? -clip_max : x);
  return int8_t(x * scale_div_clip_max);
}

__forceinline__ __host__ __device__ int8_t posfloat2int8(float x, float scale,
                                                         float clip_max) {
  x = x > clip_max ? clip_max : (x < -clip_max ? -clip_max : x);
  return int8_t(x * 2 * scale / clip_max - scale);
}

template <typename T>
__global__ void quantize_tensor_kernel(const T *input, int8_t *output,
                                       int total_count, float scale,
                                       float clip_max);

template <>
__global__ void quantize_tensor_kernel<float>(const float *input,
                                              int8_t *output, int total_count,
                                              float scale_div_clip_max,
                                              float clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i * 4 >= total_count) return;

  const float4 *input4 = reinterpret_cast<const float4 *>(input);
  int32_t *output4 = reinterpret_cast<int32_t *>(output);
  float4 inp4 = input4[i];
  int32_t out4;
  int8_t *out1 = reinterpret_cast<int8_t *>(&out4);
  out1[0] = float2int8(inp4.x, scale_div_clip_max, clip_max);
  out1[1] = float2int8(inp4.y, scale_div_clip_max, clip_max);
  out1[2] = float2int8(inp4.z, scale_div_clip_max, clip_max);
  out1[3] = float2int8(inp4.w, scale_div_clip_max, clip_max);
  output4[i] = out4;
}

template <>
__global__ void quantize_tensor_kernel<__half>(const __half *input,
                                               int8_t *output, int total_count,
                                               float scale_div_clip_max,
                                               float clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i * 8 >= total_count) return;

  const float4 *input4 = reinterpret_cast<const float4 *>(input);
  int64_t *output4 = reinterpret_cast<int64_t *>(output);
  float4 inp4 = input4[i];
  int64_t out8;
  __half *inp_h = reinterpret_cast<__half *>(&inp4);
  int8_t *out1 = reinterpret_cast<int8_t *>(&out8);
#pragma unroll
  for (uint j = 0; j < 8; ++j) {
    out1[j] = float2int8(__half2float(inp_h[j]), scale_div_clip_max, clip_max);
  }
  output4[i] = out8;
}

template <>
void launch_quantize_tensor<float>(const float *input, int8_t *output,
                                   int total_count, float scale, float clip_max,
                                   hipStream_t &stream) {
  int grid_dim = total_count >> 12;
  quantize_tensor_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      input, output, total_count, scale / clip_max, clip_max);
}

template <>
void launch_quantize_tensor<__half>(const __half *input, int8_t *output,
                                    int total_count, float scale,
                                    float clip_max, hipStream_t &stream) {
  int grid_dim = total_count >> 13;
  quantize_tensor_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      input, output, total_count, scale / clip_max, clip_max);
}

template <typename T>
__global__ void dequantize_tensor_kernel(const int32_t *input, T *output,
                                         int total_count,
                                         float scale_div_clip_max,
                                         float clip_max);

template <>
__global__ void dequantize_tensor_kernel<float>(const int32_t *input,
                                                float *output, int total_count,
                                                float scale_div_clip_max,
                                                float clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i * 4 >= total_count) return;

  const int4 *input4 = reinterpret_cast<const int4 *>(input);
  float4 *output4 = reinterpret_cast<float4 *>(output);
  int4 inp4 = input4[i];
  float4 out4;
  out4.x = float(inp4.x) / scale_div_clip_max;
  out4.y = float(inp4.y) / scale_div_clip_max;
  out4.z = float(inp4.z) / scale_div_clip_max;
  out4.w = float(inp4.w) / scale_div_clip_max;
  output4[i] = out4;
}

template <>
__global__ void dequantize_tensor_kernel<__half>(const int32_t *input,
                                                 __half *output,
                                                 int total_count,
                                                 float scale_div_clip_max,
                                                 float clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i * 8 >= total_count) return;

  const long4 *input4 = reinterpret_cast<const long4 *>(input);
  float4 *output4 = reinterpret_cast<float4 *>(output);
  long4 inp4 = input4[i];
  float4 out4;
  int32_t *inp1 = reinterpret_cast<int32_t *>(&inp4);
  __half *out1 = reinterpret_cast<__half *>(&out4);
#pragma unroll
  for (uint j = 0; j < 8; ++j) {
    out1[j] = __float2half(float(inp1[j]) / scale_div_clip_max);
  }
  output4[i] = out4;
}

template <>
void launch_dequantize_tensor<float>(const int32_t *input, float *output,
                                     int total_count, float scale,
                                     float clip_max, hipStream_t &stream) {
  int grid_dim = total_count >> 12;
  dequantize_tensor_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      input, output, total_count, scale / clip_max, clip_max);
}

template <>
void launch_dequantize_tensor<__half>(const int32_t *input, __half *output,
                                      int total_count, float scale,
                                      float clip_max, hipStream_t &stream) {
  int grid_dim = total_count >> 13;
  dequantize_tensor_kernel<<<grid_dim + 1, 1024, 0, stream>>>(
      input, output, total_count, scale / clip_max, clip_max);
}

template <typename T>
__global__ void ker_norm_layer_int8O(T *matrix, int8_t *output, const T *scale,
                                     const T *bias, int hidden_size,
                                     float scale_div_clip_max, float clip_max) {
  uint block_start = blockIdx.x * hidden_size;
  uint start = block_start + threadIdx.x;
  uint end = block_start + hidden_size;
  float val = 0.0;
  for (uint i = start; i < end; i += blockDim.x) {
    val += matrix[i];
  }

  // step 0. compute mean
  __shared__ float s_mean;
  float reduce_res = blockReduceSum<float>(val);
  if (threadIdx.x == 0) s_mean = reduce_res / float(hidden_size);
  __syncthreads();

  // step 1. compute variance
  val = 0.0;
  for (uint i = start; i < end; i += blockDim.x) {
    float tmp = matrix[i] - s_mean;
    val += tmp * tmp;
  }
  __shared__ float s_var;
  reduce_res = blockReduceSum(val);
  if (threadIdx.x == 0)
    s_var = rsqrtf(reduce_res / float(hidden_size) + epsilon);
  __syncthreads();

  float output_f;

  // step 2. layer norm
  for (uint i = start; i < end; i += blockDim.x) {
    val = matrix[i] - s_mean;
    output_f = val * s_var * __ldg(&scale[i - block_start]) +
               __ldg(&bias[i - block_start]);
    matrix[i] = float2int8(output_f, scale_div_clip_max, clip_max);
  }
}

template <>
__global__ void ker_norm_layer_int8O<__half>(
    __half *matrix, int8_t *output, const __half *scale, const __half *bias,
    int half_hidden_size, float scale_div_clip_max, float clip_max) {
  uint block_start = blockIdx.x * half_hidden_size;
  uint start = block_start + threadIdx.x;
  uint end = blockIdx.x * half_hidden_size + half_hidden_size;
  half2 *pmatrix = (half2 *)matrix;
  char2 *poutput = (char2 *)output;
  const half2 *pscale = (const half2 *)scale;
  const half2 *pbias = (const half2 *)bias;
  float mean_dim = float(half_hidden_size) * 2.f;

  float val = 0.0;
  // step 0. compute mean
  for (uint i = start; i < end; i += blockDim.x) {
    float2 local_f2 = safe_half2_to_float2(pmatrix[i]);
    val += local_f2.x + local_f2.y;
  }
  __shared__ float s_mean;
  float reduce_res = blockReduceSum<float>(val);
  if (threadIdx.x == 0) s_mean = reduce_res / mean_dim;
  __syncthreads();

  // step 1. compute variance
  val = 0.0;
  for (uint i = start; i < end; i += blockDim.x) {
    float2 local_f2 = safe_half2_to_float2(pmatrix[i]);
    float tmpx = local_f2.x - s_mean;
    float tmpy = local_f2.y - s_mean;
    val += tmpx * tmpx + tmpy * tmpy;
  }
  __shared__ float s_var;
  reduce_res = blockReduceSum(val);
  if (threadIdx.x == 0) s_var = rsqrtf(reduce_res / mean_dim + epsilon);
  __syncthreads();

  char2 output_c2;

  // step 2. layer norm
  for (uint i = start; i < end; i += blockDim.x) {
    float2 scale_val = __half22float2(__ldg(&pscale[i - block_start]));
    float2 bias_val = __half22float2(__ldg(&pbias[i - block_start]));
    float2 local_f2 = safe_half2_to_float2(pmatrix[i]);
    local_f2.x = (local_f2.x - s_mean) * s_var * scale_val.x + bias_val.x;
    local_f2.y = (local_f2.y - s_mean) * s_var * scale_val.y + bias_val.y;
    output_c2.x = float2int8(local_f2.x, scale_div_clip_max, clip_max);
    output_c2.y = float2int8(local_f2.y, scale_div_clip_max, clip_max);
    poutput[i] = output_c2;
  }
}

template <typename T>
void ker_norm_layer_int8O_launcher(int token_num, int hidden_size,
                                   hipStream_t stream, T *matrix,
                                   int8_t *output, const T *scale,
                                   const T *bias, int max_thread_per_block,
                                   float quant_scale, float clip_max) {
  ker_norm_layer_int8O<T><<<token_num, max_thread_per_block, 0, stream>>>(
      matrix, output, scale, bias, hidden_size, quant_scale / clip_max,
      clip_max);
}

template <>
void ker_norm_layer_int8O_launcher<__half>(int token_num, int hidden_size,
                                           hipStream_t stream, __half *matrix,
                                           int8_t *output, const __half *scale,
                                           const __half *bias,
                                           int max_thread_per_block,
                                           float quant_scale, float clip_max) {
  ker_norm_layer_int8O<__half><<<token_num, max_thread_per_block, 0, stream>>>(
      matrix, output, scale, bias, hidden_size / 2, quant_scale / clip_max,
      clip_max);
}

template void ker_norm_layer_int8O_launcher<float>(
    int token_num, int hidden_size, hipStream_t stream, float *matrix,
    int8_t *output, const float *scale, const float *bias,
    int max_thread_per_block, float quant_scale, float clip_max);

template void ker_norm_layer_int8O_launcher<__half>(
    int token_num, int hidden_size, hipStream_t stream, __half *matrix,
    int8_t *output, const __half *scale, const __half *bias,
    int max_thread_per_block, float quant_scale, float clip_max);

template <typename T>
__global__ void ker_norm_layer_resual_int8O(T *input, int8_t *output,
                                            const T *scale, const T *bias,
                                            const T *residual_bias,
                                            const int hidden_size,
                                            float scale_div_clip_max,
                                            float clip_max, bool is_post_ln) {
  uint block_start = blockIdx.x * hidden_size;
  uint start = block_start + threadIdx.x;
  uint end = block_start + hidden_size;
  float val = 0.0;
  for (uint i = start; i < end; i += blockDim.x) {
    val += input[i];
  }

  // step 0. compute mean
  __shared__ float s_mean;
  float reduce_res = blockReduceSum<float>(val);
  if (threadIdx.x == 0) s_mean = reduce_res / float(hidden_size);
  __syncthreads();

  // step 1. compute variance
  val = 0.0;
  for (uint i = start; i < end; i += blockDim.x) {
    float tmp = input[i] - s_mean;
    val += tmp * tmp;
  }
  __shared__ float s_var;
  reduce_res = blockReduceSum(val);
  if (threadIdx.x == 0)
    s_var = rsqrtf(reduce_res / float(hidden_size) + epsilon);
  __syncthreads();

  float output_f;

  // step 2. layer norm
  for (uint i = start; i < end; i += blockDim.x) {
    val = input[i] - s_mean;
    output_f = val * s_var * __ldg(&scale[i - block_start]) +
               __ldg(&bias[i - block_start]);
    output[i] = float2int8(output_f, scale_div_clip_max, clip_max);
    if (is_post_ln) {
      input[i] = output_f + __ldg(&residual_bias[i - block_start]);
    } else {
      input[i] += __ldg(&residual_bias[i - block_start]);
    }
  }
}

template <>
__global__ void ker_norm_layer_resual_int8O<__half>(
    __half *input, int8_t *output, const __half *scale, const __half *bias,
    const __half *residual_bias, const int half_hidden_size,
    float scale_div_clip_max, float clip_max, bool is_post_ln) {
  uint block_start = blockIdx.x * half_hidden_size;
  uint start = block_start + threadIdx.x;
  uint end = blockIdx.x * half_hidden_size + half_hidden_size;
  half2 *pinput = (half2 *)input;
  char2 *poutput = (char2 *)output;
  const half2 *pscale = (const half2 *)scale;
  const half2 *pbias = (const half2 *)bias;
  const half2 *presidual_bias = (const half2 *)residual_bias;
  float mean_dim = float(half_hidden_size) * 2.f;

  float val = 0.0;
  // step 0. compute mean
  for (uint i = start; i < end; i += blockDim.x) {
    float2 local_f2 = safe_half2_to_float2(pinput[i]);
    val += local_f2.x + local_f2.y;
  }
  __shared__ float s_mean;
  float reduce_res = blockReduceSum<float>(val);
  if (threadIdx.x == 0) s_mean = reduce_res / mean_dim;
  __syncthreads();

  // step 1. compute variance
  val = 0.0;
  for (uint i = start; i < end; i += blockDim.x) {
    float2 local_f2 = safe_half2_to_float2(pinput[i]);
    float tmpx = local_f2.x - s_mean;
    float tmpy = local_f2.y - s_mean;
    val += tmpx * tmpx + tmpy * tmpy;
  }
  __shared__ float s_var;
  reduce_res = blockReduceSum(val);
  if (threadIdx.x == 0) s_var = rsqrtf(reduce_res / mean_dim + epsilon);
  __syncthreads();

  char2 output_c2;

  // step 2. layer norm
  for (uint i = start; i < end; i += blockDim.x) {
    float2 scale_val = __half22float2(__ldg(&pscale[i - block_start]));
    float2 bias_val = __half22float2(__ldg(&pbias[i - block_start]));
    float2 local_f2 = safe_half2_to_float2(pinput[i]);
    local_f2.x = (local_f2.x - s_mean) * s_var * scale_val.x + bias_val.x;
    local_f2.y = (local_f2.y - s_mean) * s_var * scale_val.y + bias_val.y;
    output_c2.x = float2int8(local_f2.x, scale_div_clip_max, clip_max);
    output_c2.y = float2int8(local_f2.y, scale_div_clip_max, clip_max);
    poutput[i] = output_c2;
    if (!is_post_ln) {
      local_f2 = safe_half2_to_float2(pinput[i]);
    }
    float2 residual_bias_val =
        __half22float2(__ldg(&presidual_bias[i - block_start]));
    float2 new_input_f2;
    new_input_f2.x = local_f2.x + residual_bias_val.x;
    new_input_f2.y = local_f2.y + residual_bias_val.y;
    pinput[i] = __float22half2_rn(new_input_f2);
  }
}

template <typename T>
void ker_norm_layer_resual_int8O_launcher(int token_num, int hidden_size,
                                          hipStream_t stream, T *input,
                                          int8_t *output, const T *scale,
                                          const T *bias, const T *residual_bias,
                                          const int max_thread_per_block,
                                          float quant_scale, float clip_max,
                                          bool is_post_ln) {
  ker_norm_layer_resual_int8O<T>
      <<<token_num, max_thread_per_block, 0, stream>>>(
          input, output, scale, bias, residual_bias, hidden_size,
          quant_scale / clip_max, clip_max, is_post_ln);
}

template <>
void ker_norm_layer_resual_int8O_launcher<__half>(
    int token_num, int hidden_size, hipStream_t stream, __half *input,
    int8_t *output, const __half *scale, const __half *bias,
    const __half *residual_bias, const int max_thread_per_block,
    float quant_scale, float clip_max, bool is_post_ln) {
  ker_norm_layer_resual_int8O<__half>
      <<<token_num, max_thread_per_block, 0, stream>>>(
          input, output, scale, bias, residual_bias, hidden_size / 2,
          quant_scale / clip_max, clip_max, is_post_ln);
}

template void ker_norm_layer_resual_int8O_launcher<float>(
    int token_num, int hidden_size, hipStream_t stream, float *input,
    int8_t *output, const float *scale, const float *bias,
    const float *residual_bias, const int max_thread_per_block,
    float quant_scale, float clip_max, bool is_post_ln);

template void ker_norm_layer_resual_int8O_launcher<__half>(
    int token_num, int hidden_size, hipStream_t stream, __half *input,
    int8_t *output, const __half *scale, const __half *bias,
    const __half *residual_bias, const int max_thread_per_block,
    float quant_scale, float clip_max, bool is_post_ln);

template <typename T>
__global__ void ker_bias_gelu_int32I_int8O(int32_t *input, int8_t *output,
                                           const T *bias, int total_count,
                                           int feature_dim,
                                           float in_scale_div_clip_max,
                                           float out_scale_div_clip_max,
                                           float out_clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 4 >= total_count) return;

  char4 *out4 = reinterpret_cast<char4 *>(output);
  const int4 *data4 = reinterpret_cast<const int4 *>(input);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  int bias_i = i % (feature_dim >> 2);

  const int4 input4 = data4[i];
  const float4 b4 = __ldg(&bias4[bias_i]);
  float4 output4;

  output4.x = gelu<float>(float(input4.x) / in_scale_div_clip_max + b4.x);
  output4.y = gelu<float>(float(input4.y) / in_scale_div_clip_max + b4.y);
  output4.z = gelu<float>(float(input4.z) / in_scale_div_clip_max + b4.z);
  output4.w = gelu<float>(float(input4.w) / in_scale_div_clip_max + b4.w);

  char4 out_i4;
  out_i4.x = float2int8(output4.x, out_scale_div_clip_max, out_clip_max);
  out_i4.y = float2int8(output4.y, out_scale_div_clip_max, out_clip_max);
  out_i4.z = float2int8(output4.z, out_scale_div_clip_max, out_clip_max);
  out_i4.w = float2int8(output4.w, out_scale_div_clip_max, out_clip_max);
  out4[i] = out_i4;
}

/* fp16 version */
template <>
__global__ void ker_bias_gelu_int32I_int8O<__half>(
    int32_t *input, int8_t *output, const __half *bias, int total_count,
    int feature_dim, float in_scale_div_clip_max, float out_scale_div_clip_max,
    float out_clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 8 >= total_count) return;

  const long4 *vals_long4 = reinterpret_cast<const long4 *>(input);
  int64_t *outs_i8 = reinterpret_cast<int64_t *>(output);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);

  int bias_i = i % (feature_dim >> 3);
  long4 val_long4 = vals_long4[i];
  int32_t *val1 = reinterpret_cast<int32_t *>(&val_long4);
  const float4 b4 = __ldg(&bias4[bias_i]);
  const __half *b_half = reinterpret_cast<const __half *>(&b4);
  int64_t out_i8;
  int8_t *out_i1 = reinterpret_cast<int8_t *>(&out_i8);

#pragma unroll
  for (uint j = 0; j < 8; ++j) {
    float out_f;
    out_f = gelu<float>(float(val1[j]) / in_scale_div_clip_max +
                        __half2float(b_half[j]));
    out_i1[j] = float2int8(out_f, out_scale_div_clip_max, out_clip_max);
  }
  outs_i8[i] = out_i8;
}

template <typename T>
void ker_bias_gelu_int32I_int8O_launcher(int batch_token_num,
                                         hipStream_t stream, int32_t *input,
                                         int8_t *output, const T *bias,
                                         int feature_dim, float in_scale,
                                         float in_clip_max, float out_scale,
                                         float out_clip_max) {
  int total_count = batch_token_num * feature_dim;
  int grid_dim = total_count >> 10;
  ker_bias_gelu_int32I_int8O<T><<<grid_dim + 1, 256, 0, stream>>>(
      input, output, bias, total_count, feature_dim, in_scale / in_clip_max,
      out_scale / out_clip_max, out_clip_max);
}

template <>
void ker_bias_gelu_int32I_int8O_launcher<__half>(
    int batch_token_num, hipStream_t stream, int32_t *input, int8_t *output,
    const __half *bias, int feature_dim, float in_scale, float in_clip_max,
    float out_scale, float out_clip_max) {
  int total_count = batch_token_num * feature_dim;
  int grid_dim = total_count >> 11;
  ker_bias_gelu_int32I_int8O<__half><<<grid_dim + 1, 256, 0, stream>>>(
      input, output, bias, total_count, feature_dim, in_scale / in_clip_max,
      out_scale / out_clip_max, out_clip_max);
}

template void ker_bias_gelu_int32I_int8O_launcher<float>(
    int batch_token_num, hipStream_t stream, int32_t *input, int8_t *output,
    const float *bias, int feature_dim, float in_scale, float in_clip_max,
    float out_scale, float out_clip_max);

template void ker_bias_gelu_int32I_int8O_launcher<__half>(
    int batch_token_num, hipStream_t stream, int32_t *input, int8_t *output,
    const __half *bias, int feature_dim, float in_scale, float in_clip_max,
    float out_scale, float out_clip_max);

template <typename T>
__global__ void ker_bias_relu_int32I_int8O(int32_t *input, int8_t *output,
                                           const T *bias, int total_count,
                                           int feature_dim,
                                           float in_scale_div_clip_max,
                                           float out_scale_div_clip_max,
                                           float out_clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 4 >= total_count) return;

  char4 *out4 = reinterpret_cast<char4 *>(output);
  const int4 *data4 = reinterpret_cast<const int4 *>(input);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);
  int bias_i = i % (feature_dim >> 2);

  const int4 input4 = data4[i];
  const float4 b4 = __ldg(&bias4[bias_i]);
  float4 output4;

  output4.x = max(float(input4.x) / in_scale_div_clip_max + b4.x, (T)0.f);
  output4.y = max(float(input4.y) / in_scale_div_clip_max + b4.y, (T)0.f);
  output4.z = max(float(input4.z) / in_scale_div_clip_max + b4.z, (T)0.f);
  output4.w = max(float(input4.w) / in_scale_div_clip_max + b4.w, (T)0.f);

  char4 out_i4;
  out_i4.x = float2int8(output4.x, out_scale_div_clip_max, out_clip_max);
  out_i4.y = float2int8(output4.y, out_scale_div_clip_max, out_clip_max);
  out_i4.z = float2int8(output4.z, out_scale_div_clip_max, out_clip_max);
  out_i4.w = float2int8(output4.w, out_scale_div_clip_max, out_clip_max);
  out4[i] = out_i4;
}

/* fp16 version */
template <>
__global__ void ker_bias_relu_int32I_int8O<__half>(
    int32_t *input, int8_t *output, const __half *bias, int total_count,
    int feature_dim, float in_scale_div_clip_max, float out_scale_div_clip_max,
    float out_clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 8 >= total_count) return;

  const long4 *vals_long4 = reinterpret_cast<const long4 *>(input);
  int64_t *outs_i8 = reinterpret_cast<int64_t *>(output);
  const float4 *bias4 = reinterpret_cast<const float4 *>(bias);

  int bias_i = i % (feature_dim >> 3);
  long4 val_long4 = vals_long4[i];
  int32_t *val1 = reinterpret_cast<int32_t *>(&val_long4);
  const float4 b4 = __ldg(&bias4[bias_i]);
  const __half *b_half = reinterpret_cast<const __half *>(&b4);
  int64_t out_i8;
  int8_t *out_i1 = reinterpret_cast<int8_t *>(&out_i8);

#pragma unroll
  for (uint j = 0; j < 8; ++j) {
    float out_f;
    out_f =
        max(float(val1[j]) / in_scale_div_clip_max + __half2float(b_half[j]),
            (float)0.f);
    out_i1[j] = float2int8(out_f, out_scale_div_clip_max, out_clip_max);
  }
  outs_i8[i] = out_i8;
}

template <typename T>
void ker_bias_relu_int32I_int8O_launcher(int batch_token_num,
                                         hipStream_t stream, int32_t *input,
                                         int8_t *output, const T *bias,
                                         int feature_dim, float in_scale,
                                         float in_clip_max, float out_scale,
                                         float out_clip_max) {
  int total_count = batch_token_num * feature_dim;
  int grid_dim = total_count >> 10;
  ker_bias_relu_int32I_int8O<T><<<grid_dim + 1, 256, 0, stream>>>(
      input, output, bias, total_count, feature_dim, in_scale / in_clip_max,
      out_scale / out_clip_max, out_clip_max);
}

template <>
void ker_bias_relu_int32I_int8O_launcher<__half>(
    int batch_token_num, hipStream_t stream, int32_t *input, int8_t *output,
    const __half *bias, int feature_dim, float in_scale, float in_clip_max,
    float out_scale, float out_clip_max) {
  int total_count = batch_token_num * feature_dim;
  int grid_dim = total_count >> 11;
  ker_bias_relu_int32I_int8O<__half><<<grid_dim + 1, 256, 0, stream>>>(
      input, output, bias, total_count, feature_dim, in_scale / in_clip_max,
      out_scale / out_clip_max, out_clip_max);
}

template void ker_bias_relu_int32I_int8O_launcher<float>(
    int batch_token_num, hipStream_t stream, int32_t *input, int8_t *output,
    const float *bias, int feature_dim, float in_scale, float in_clip_max,
    float out_scale, float out_clip_max);

template void ker_bias_relu_int32I_int8O_launcher<__half>(
    int batch_token_num, hipStream_t stream, int32_t *input, int8_t *output,
    const __half *bias, int feature_dim, float in_scale, float in_clip_max,
    float out_scale, float out_clip_max);

template <typename T>
__global__ void ker_residual_int32I(int32_t *input, T *output, int total_count,
                                    float scale_div_clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 4 >= total_count) return;

  float4 *out4 = reinterpret_cast<float4 *>(output);
  const int4 *data4 = reinterpret_cast<const int4 *>(input);
  const int4 input4 = data4[i];
  float4 output4 = out4[i];

  output4.x += float(input4.x) / scale_div_clip_max;
  output4.y += float(input4.y) / scale_div_clip_max;
  output4.z += float(input4.z) / scale_div_clip_max;
  output4.w += float(input4.w) / scale_div_clip_max;

  out4[i] = output4;
}

/* fp16 version */
template <>
__global__ void ker_residual_int32I<__half>(int32_t *input, __half *output,
                                            int total_count,
                                            float scale_div_clip_max) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i * 8 >= total_count) return;

  const long4 *vals_long4 = reinterpret_cast<const long4 *>(input);
  float4 *outs_h8 = reinterpret_cast<float4 *>(output);
  long4 val_long4 = vals_long4[i];
  int32_t *val1 = reinterpret_cast<int32_t *>(&val_long4);
  float4 out_h8 = outs_h8[i];
  __half *out_h1 = reinterpret_cast<__half *>(&out_h8);

#pragma unroll
  for (uint j = 0; j < 8; ++j) {
    out_h1[j] =
        __hadd(out_h1[j], __float2half(float(val1[j]) / scale_div_clip_max));
  }
  outs_h8[i] = out_h8;
}

template <typename T>
void ker_residual_int32I_launcher(int32_t *input, T *output, int total_ele_num,
                                  float quant_scale, float clip_max,
                                  hipStream_t stream) {
  int grid_dim = total_ele_num >> 10;
  ker_residual_int32I<T><<<grid_dim + 1, 256, 0, stream>>>(
      input, output, total_ele_num, quant_scale / clip_max);
}

template <>
void ker_residual_int32I_launcher<__half>(int32_t *input, __half *output,
                                          int total_ele_num, float quant_scale,
                                          float clip_max, hipStream_t stream) {
  int grid_dim = total_ele_num >> 11;
  ker_residual_int32I<__half><<<grid_dim + 1, 256, 0, stream>>>(
      input, output, total_ele_num, quant_scale / clip_max);
}

template void ker_residual_int32I_launcher<float>(int32_t *input, float *output,
                                                  int total_ele_num,
                                                  float quant_scale,
                                                  float clip_max,
                                                  hipStream_t stream);

template void ker_residual_int32I_launcher<__half>(
    int32_t *input, __half *output, int total_ele_num, float quant_scale,
    float clip_max, hipStream_t stream);

template <typename T>
__global__ void ker_arrange_encself_qkv_int32I(const int32_t *ori_qkv,
                                               const T *qkv_bias, T *new_qkv,
                                               int max_batch_dim,
                                               int batch_seq_len,
                                               int dim_per_head, int head_num,
                                               float scale_div_clip_max) {
  int hidden_size = dim_per_head * head_num;
  int batch_id = blockIdx.x / batch_seq_len;
  int token_id = blockIdx.x % batch_seq_len;
  int qkv_offset = max_batch_dim * blockIdx.y;
  for (std::size_t i = threadIdx.x; i < hidden_size; i += blockDim.x) {
    int head_id = i / dim_per_head;
    int dim_id = i % dim_per_head;
    int target_id = targetid_4dim(batch_id, head_id, token_id, dim_id, head_num,
                                  batch_seq_len, dim_per_head);
    new_qkv[qkv_offset + target_id] =
        float(
            ori_qkv[(blockIdx.x * gridDim.y + blockIdx.y) * hidden_size + i]) /
            scale_div_clip_max +
        __ldg(&qkv_bias[blockIdx.y * hidden_size + i]);
  }
}

template <>
__global__ void ker_arrange_encself_qkv_int32I<__half>(
    const int32_t *ori_qkv, const __half *qkv_bias, __half *new_qkv,
    int max_batch_dim, int batch_seq_len, int dim_per_head, int head_num,
    float scale_div_clip_max) {
  int hidden_size = dim_per_head * head_num;
  int batch_id = blockIdx.x / batch_seq_len;
  int token_id = blockIdx.x % batch_seq_len;
  for (std::size_t i = threadIdx.x; i < hidden_size; i += blockDim.x) {
    int head_id = i / dim_per_head;
    int dim_id = i % dim_per_head;
    int qkv_offset = max_batch_dim * blockIdx.y;
    int target_id = targetid_4dim(batch_id, head_id, token_id, dim_id, head_num,
                                  batch_seq_len, dim_per_head);

    const int2 *p_ori_qkv = (const int2 *)ori_qkv;
    const half2 *p_bias = (const half2 *)qkv_bias;
    half2 *p_new_qkv = (half2 *)new_qkv;
    int2 ori_qkv_i2 =
        p_ori_qkv[(blockIdx.x * gridDim.y + blockIdx.y) * hidden_size + i];
    half2 ori_qkv_h2;
    ori_qkv_h2.x = __float2half(float(ori_qkv_i2.x) / scale_div_clip_max);
    ori_qkv_h2.y = __float2half(float(ori_qkv_i2.y) / scale_div_clip_max);
    p_new_qkv[qkv_offset + target_id] =
        __hadd2(ori_qkv_h2, __ldg(&p_bias[blockIdx.y * hidden_size + i]));
  }
}

template <typename T>
void ker_arrange_encself_qkv_int32I_launcher(
    int batch_token_num, int hidden_size, hipStream_t stream,
    const int32_t *ori_qkv, const T *qkv_bias, T *new_qkv, int max_batch_dim,
    int batch_seq_len, int dim_per_head, int head_num, int max_thread_per_block,
    float quant_scale, float clip_max) {
  ker_arrange_encself_qkv_int32I<T>
      <<<dim3(batch_token_num, 3), max_thread_per_block, 0, stream>>>(
          ori_qkv, qkv_bias, new_qkv, max_batch_dim, batch_seq_len,
          dim_per_head, head_num, quant_scale / clip_max);
}

template <>
void ker_arrange_encself_qkv_int32I_launcher<__half>(
    int batch_token_num, int hidden_size, hipStream_t stream,
    const int32_t *ori_qkv, const __half *qkv_bias, __half *new_qkv,
    int max_batch_dim, int batch_seq_len, int dim_per_head, int head_num,
    int max_thread_per_block, float quant_scale, float clip_max) {
  ker_arrange_encself_qkv_int32I<__half>
      <<<dim3(batch_token_num, 3), max_thread_per_block, 0, stream>>>(
          ori_qkv, qkv_bias, new_qkv, max_batch_dim / 2, batch_seq_len,
          dim_per_head / 2, head_num, quant_scale / clip_max);
}

template void ker_arrange_encself_qkv_int32I_launcher<float>(
    int batch_token_num, int hidden_size, hipStream_t stream,
    const int32_t *ori_qkv, const float *qkv_bias, float *new_qkv,
    int max_batch_dim, int batch_seq_len, int dim_per_head, int head_num,
    int max_thread_per_block, float quant_scale, float clip_max);

template void ker_arrange_encself_qkv_int32I_launcher<__half>(
    int batch_token_num, int hidden_size, hipStream_t stream,
    const int32_t *ori_qkv, const __half *qkv_bias, __half *new_qkv,
    int max_batch_dim, int batch_seq_len, int dim_per_head, int head_num,
    int max_thread_per_block, float quant_scale, float clip_max);

template <typename T>
__global__ void ker_arrange_atten_output_int8O(const T *ori_q, int8_t *new_q,
                                               int beam_size, int dim_per_head,
                                               int head_num,
                                               float scale_div_clip_max,
                                               float clip_max) {
  int hidden_size = dim_per_head * head_num;
  int batch_id = blockIdx.x / beam_size;
  // note, for encoder, beam_id is token_id; for decoder, beam_id is beam_id
  int beam_id = blockIdx.x % beam_size;
  for (std::size_t i = threadIdx.x; i < hidden_size; i += blockDim.x) {
    int head_id = i / dim_per_head;
    int dim_id = i % dim_per_head;
    new_q[blockIdx.x * hidden_size + i] =
        float2int8(ori_q[targetid_4dim(batch_id, head_id, beam_id, dim_id,
                                       head_num, beam_size, dim_per_head)],
                   scale_div_clip_max, clip_max);
  }
}

template <>
__global__ void ker_arrange_atten_output_int8O<__half>(
    const __half *ori_q, int8_t *new_q, int beam_size, int dim_per_head,
    int head_num, float scale_div_clip_max, float clip_max) {
  int batch_id = blockIdx.x / beam_size;
  // note, for encoder, beam_id is token_id; for decoder, beam_id is beam_id
  int beam_id = blockIdx.x % beam_size;
  int half_hidden_size = dim_per_head * head_num;
  for (std::size_t i = threadIdx.x; i < half_hidden_size; i += blockDim.x) {
    int head_id = i / dim_per_head;
    int dim_id = i % dim_per_head;
    const half2 *p_ori_q = (const half2 *)ori_q;
    half2 v_ori_q;
    char2 *p_new_q = (char2 *)new_q;
    char2 v_new_q;
    v_ori_q = p_ori_q[targetid_4dim(batch_id, head_id, beam_id, dim_id,
                                    head_num, beam_size, dim_per_head)];
    v_new_q.x = float2int8(float(v_ori_q.x), scale_div_clip_max, clip_max);
    v_new_q.y = float2int8(float(v_ori_q.y), scale_div_clip_max, clip_max);
    p_new_q[blockIdx.x * half_hidden_size + i] = v_new_q;
  }
}

template <typename T>
void ker_arrange_atten_output_int8O_launcher(
    int batch_token_num, int hidden_size, hipStream_t stream, const T *ori_q,
    int8_t *new_q, int beam_size, int dim_per_head, int head_num,
    int max_thread_per_block, float quant_scale, float clip_max) {
  ker_arrange_atten_output_int8O<T>
      <<<batch_token_num, max_thread_per_block, 0, stream>>>(
          ori_q, new_q, beam_size, dim_per_head, head_num,
          quant_scale / clip_max, clip_max);
}

template <>
void ker_arrange_atten_output_int8O_launcher<__half>(
    int batch_token_num, int hidden_size, hipStream_t stream,
    const __half *ori_q, int8_t *new_q, int beam_size, int dim_per_head,
    int head_num, int max_thread_per_block, float quant_scale, float clip_max) {
  ker_arrange_atten_output_int8O<__half>
      <<<batch_token_num, max_thread_per_block, 0, stream>>>(
          ori_q, new_q, beam_size, dim_per_head / 2, head_num,
          quant_scale / clip_max, clip_max);
}

template void ker_arrange_atten_output_int8O_launcher<float>(
    int batch_token_num, int hidden_size, hipStream_t stream,
    const float *ori_q, int8_t *new_q, int beam_size, int dim_per_head,
    int head_num, int max_thread_per_block, float quant_scale, float clip_max);

template void ker_arrange_atten_output_int8O_launcher<__half>(
    int batch_token_num, int hidden_size, hipStream_t stream,
    const __half *ori_q, int8_t *new_q, int beam_size, int dim_per_head,
    int head_num, int max_thread_per_block, float quant_scale, float clip_max);

template <typename T>
__global__ void ker_arrange_decself_qkv_int32I(const int32_t *ori_qkv,
                                               const T *qkv_bias, T *new_q,
                                               T *new_k, T *new_v, int head_num,
                                               int dim_per_head, int max_step,
                                               int step_id,
                                               float scale_div_clip_max) {
  int hidden_size = dim_per_head * head_num;
  for (std::size_t i = threadIdx.x; i < hidden_size; i += blockDim.x) {
    // blockdim is equal to hidden_size
    T val =
        float(
            ori_qkv[(blockIdx.x * gridDim.y + blockIdx.y) * hidden_size + i]) /
            scale_div_clip_max +
        __ldg(&qkv_bias[blockIdx.y * hidden_size + i]);
    int seq_id =
        blockIdx.x;  // obvious， seq_id = batch_id * beam_size + beam_id
    if (blockIdx.y == 0) {
      // for query
      new_q[seq_id * hidden_size + i] = val;
      return;
    }
    int head_id = i / dim_per_head;
    int dim_id = i % dim_per_head;
    int target_id = targetid_4dim(seq_id, head_id, step_id, dim_id, head_num,
                                  max_step, dim_per_head);
    if (blockIdx.y == 1) {
      // for key
      new_k[target_id] = val;
    } else {
      // for value
      new_v[target_id] = val;
    }
  }
}

template <>
__global__ void ker_arrange_decself_qkv_int32I<__half>(
    const int32_t *ori_qkv, const __half *qkv_bias, __half *new_q,
    __half *new_k, __half *new_v, int head_num, int dim_per_head, int max_step,
    int step_id, float scale_div_clip_max) {
  int half_hidden_size = dim_per_head * head_num;
  const int2 *p_qkv = (const int2 *)ori_qkv;
  const half2 *p_bias = (const half2 *)qkv_bias;
  int2 v_ori_qkv;
  half2 ori_qkv_h2;
  for (std::size_t i = threadIdx.x; i < half_hidden_size; i += blockDim.x) {
    v_ori_qkv =
        p_qkv[(blockIdx.x * gridDim.y + blockIdx.y) * half_hidden_size + i];
    ori_qkv_h2.x = __float2half(float(v_ori_qkv.x) / scale_div_clip_max);
    ori_qkv_h2.y = __float2half(float(v_ori_qkv.y) / scale_div_clip_max);
    half2 val =
        __hadd2(ori_qkv_h2, __ldg(&p_bias[blockIdx.y * half_hidden_size + i]));
    // obvious，seq_id = batch_id * beam_size + beam_id
    int seq_id = blockIdx.x;
    if (blockIdx.y == 0) {
      // for query
      ((half2 *)new_q)[seq_id * half_hidden_size + i] = val;
      return;
    }
    int head_id = i / dim_per_head;
    int dim_id = i % dim_per_head;
    int target_id = targetid_4dim(seq_id, head_id, step_id, dim_id, head_num,
                                  max_step, dim_per_head);
    if (blockIdx.y == 1) {
      // for key
      ((half2 *)new_k)[target_id] = val;
    } else {
      // for value
      ((half2 *)new_v)[target_id] = val;
    }
  }
}

template <typename T>
void ker_arrange_decself_qkv_int32I_launcher(
    int step_token_num, int hidden_size, hipStream_t stream,
    const int32_t *ori_qkv, const T *qkv_bias, T *new_q, T *new_k, T *new_v,
    int head_num, int dim_per_head, int max_step, int step_id,
    int max_thread_per_block, float quant_scale, float clip_max) {
  ker_arrange_decself_qkv_int32I<T>
      <<<dim3(step_token_num, 3), max_thread_per_block, 0, stream>>>(
          ori_qkv, qkv_bias, new_q, new_k, new_v, head_num, dim_per_head,
          max_step, step_id, quant_scale / clip_max);
}

template <>
void ker_arrange_decself_qkv_int32I_launcher<__half>(
    int step_token_num, int hidden_size, hipStream_t stream,
    const int32_t *ori_qkv, const __half *qkv_bias, __half *new_q,
    __half *new_k, __half *new_v, int head_num, int dim_per_head, int max_step,
    int step_id, int max_thread_per_block, float quant_scale, float clip_max) {
  ker_arrange_decself_qkv_int32I<__half>
      <<<dim3(step_token_num, 3), max_thread_per_block, 0, stream>>>(
          ori_qkv, qkv_bias, new_q, new_k, new_v, head_num, dim_per_head / 2,
          max_step, step_id, quant_scale / clip_max);
}

template void ker_arrange_decself_qkv_int32I_launcher<float>(
    int step_token_num, int hidden_size, hipStream_t stream,
    const int32_t *ori_qkv, const float *qkv_bias, float *new_q, float *new_k,
    float *new_v, int head_num, int dim_per_head, int max_step, int step_id,
    int max_thread_per_block, float quant_scale, float clip_max);

template void ker_arrange_decself_qkv_int32I_launcher<__half>(
    int step_token_num, int hidden_size, hipStream_t stream,
    const int32_t *ori_qkv, const __half *qkv_bias, __half *new_q,
    __half *new_k, __half *new_v, int head_num, int dim_per_head, int max_step,
    int step_id, int max_thread_per_block, float quant_scale, float clip_max);

template <typename T>
__global__ void ker_arrange_encdec_q_int32I(const int32_t *ori_q,
                                            const T *q_bias, T *new_q,
                                            int beam_size, int dim_per_head,
                                            int head_num,
                                            float scale_div_clip_max) {
  int hidden_size = dim_per_head * head_num;
  for (std::size_t i = threadIdx.x; i < hidden_size; i += blockDim.x) {
    T val = float(ori_q[blockIdx.x * hidden_size + i]) / scale_div_clip_max +
            __ldg(&q_bias[i]);
    int batch_id = blockIdx.x / beam_size;
    int beam_id = blockIdx.x % beam_size;
    int head_id = i / dim_per_head;
    int dim_id = i % dim_per_head;
    new_q[targetid_4dim(batch_id, head_id, beam_id, dim_id, head_num, beam_size,
                        dim_per_head)] = val;
  }
}

template <>
__global__ void ker_arrange_encdec_q_int32I<__half>(
    const int32_t *ori_q, const __half *q_bias, __half *new_q, int beam_size,
    int dim_per_head, int head_num, float scale_div_clip_max) {
  int half_hidden_size = dim_per_head * head_num;
  for (std::size_t i = threadIdx.x; i < half_hidden_size; i += blockDim.x) {
    const int2 *p_q = (const int2 *)ori_q;
    int2 p_q_i2 = p_q[blockIdx.x * half_hidden_size + i];
    half2 p_q_h2;
    p_q_h2.x = __float2half(float(p_q_i2.x) / scale_div_clip_max);
    p_q_h2.y = __float2half(float(p_q_i2.y) / scale_div_clip_max);
    const half2 *p_bias = (const half2 *)q_bias;
    half2 val = __hadd2(p_q_h2, __ldg(&p_bias[i]));
    int batch_id = blockIdx.x / beam_size;
    int beam_id = blockIdx.x % beam_size;
    int head_id = i / dim_per_head;
    int dim_id = i % dim_per_head;
    ((half2 *)new_q)[targetid_4dim(batch_id, head_id, beam_id, dim_id, head_num,
                                   beam_size, dim_per_head)] = val;
  }
}

template <typename T>
void ker_arrange_encdec_q_int32I_launcher(int step_token_num, int hidden_size,
                                          hipStream_t stream,
                                          const int32_t *ori_q, const T *q_bias,
                                          T *new_q, int beam_size,
                                          int dim_per_head, int head_num,
                                          int max_thread_per_block,
                                          float quant_scale, float clip_max) {
  ker_arrange_encdec_q_int32I<T>
      <<<step_token_num, max_thread_per_block, 0, stream>>>(
          ori_q, q_bias, new_q, beam_size, dim_per_head, head_num,
          quant_scale / clip_max);
}

template <>
void ker_arrange_encdec_q_int32I_launcher<__half>(
    int step_token_num, int hidden_size, hipStream_t stream,
    const int32_t *ori_q, const __half *q_bias, __half *new_q, int beam_size,
    int dim_per_head, int head_num, int max_thread_per_block, float quant_scale,
    float clip_max) {
  ker_arrange_encdec_q_int32I<__half>
      <<<step_token_num, max_thread_per_block, 0, stream>>>(
          ori_q, q_bias, new_q, beam_size, dim_per_head / 2, head_num,
          quant_scale / clip_max);
}

template void ker_arrange_encdec_q_int32I_launcher<float>(
    int step_token_num, int hidden_size, hipStream_t stream,
    const int32_t *ori_q, const float *q_bias, float *new_q, int beam_size,
    int dim_per_head, int head_num, int max_thread_per_block, float quant_scale,
    float clip_max);

template void ker_arrange_encdec_q_int32I_launcher<__half>(
    int step_token_num, int hidden_size, hipStream_t stream,
    const int32_t *ori_q, const __half *q_bias, __half *new_q, int beam_size,
    int dim_per_head, int head_num, int max_thread_per_block, float quant_scale,
    float clip_max);

}  // namespace cuda
}  // namespace lightseq
